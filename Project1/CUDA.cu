#include "hip/hip_runtime.h"
#include <math.h>

void SimpleSummator(double* a, double* b, double* c, int length){
	for (int i = 0; i < length; i++){
		c[i] = sinf(a[i]) + sinf(b[i]);
	 }
}

__global__ void CUDASummator(double* a, double* b, double* c){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	c[i] = sinf(a[i]) + sinf(b[i]);	
}

__global__ void CUDASinusator(double* a, double* result){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	result[i] = sinf(a[i]);	
}

__global__ void RangeSummator(double* a, double* b, double* c, int bottomB){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	c[i] = sinf(a[i]);	
}

int GPU_Sinusator(double* a, double* result, int length){
	double* dev_a;	
	double* dev_result;

	hipMalloc((void**)&dev_a, length * sizeof(double));	
	hipMalloc((void**)&dev_result, length * sizeof(double));		

	hipMemcpy(dev_a, a, length * sizeof(double), hipMemcpyHostToDevice);	
	CUDASinusator<<<100,length/100>>>(dev_a,dev_result);
	hipMemcpy(result,dev_result,length * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(dev_a);	
	hipFree(dev_result);
	return 9999;
}

int count = 100;
int GPU_Summator(double* a, double* b, double* c, int length, int type){
	if (type == 1) {
		SimpleSummator(a,b,c,length);
		return 11111;
	}
	if (type == 2) {
		double* dev_a;
		double* dev_b;
		double* dev_c;
		
		float elapsed=0;
		hipEvent_t start, stop;

		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start, 0);

		hipMalloc((void**)&dev_a, length * sizeof(double));
		hipMalloc((void**)&dev_b, length * sizeof(double)); 
		hipMalloc((void**)&dev_c, length * sizeof(double));		

		hipMemcpy(dev_a, a, length * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(dev_b, b, length * sizeof(double), hipMemcpyHostToDevice);
		CUDASummator<<<count,length/count>>>(dev_a,dev_b,dev_c);
		hipMemcpy(c,dev_c,length * sizeof(double), hipMemcpyDeviceToHost);

		hipFree(dev_a);
		hipFree(dev_b);
		hipFree(dev_c);

		hipEventRecord(stop, 0);
		hipEventSynchronize (stop);

		hipEventElapsedTime(&elapsed, start, stop);

		hipEventDestroy(start);
		hipEventDestroy(stop);

		return (int)(elapsed*1000);
	}
	int repeatC = 4;
	float el;
	if (type == 3){
		float bestT = 10000.0;
		float bestC = 10;
		for (int i = 10; i < 1000; i+=10){
			el = 0.0;
			for (int r = 0; r < repeatC; r++){
				double* dev_a;
				double* dev_b;
				double* dev_c;
		
				float elapsed=0;
				hipEvent_t start, stop;

				hipEventCreate(&start);
				hipEventCreate(&stop);

				hipEventRecord(start, 0);

				hipMalloc((void**)&dev_a, length * sizeof(double));
				hipMalloc((void**)&dev_b, length * sizeof(double)); 
				hipMalloc((void**)&dev_c, length * sizeof(double));		

				hipMemcpy(dev_a, a, length * sizeof(double), hipMemcpyHostToDevice);
				hipMemcpy(dev_b, b, length * sizeof(double), hipMemcpyHostToDevice);
				CUDASummator<<<i,length/i>>>(dev_a,dev_b,dev_c);
				hipMemcpy(c,dev_c,length * sizeof(double), hipMemcpyDeviceToHost);

				hipFree(dev_a);
				hipFree(dev_b);
				hipFree(dev_c);

				hipEventRecord(stop, 0);
				hipEventSynchronize (stop);

				hipEventElapsedTime(&elapsed, start, stop);

				hipEventDestroy(start);
				hipEventDestroy(stop);
				el += elapsed;
			}
			if (el/repeatC < bestT) {
				bestT = el/repeatC;
				bestC = i;
			}
		}
		return bestC;
	}
	return 99999;
};